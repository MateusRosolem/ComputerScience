#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>


#define THREADS_PER_BLOCK 512
#define NUM_RUNS 10

#define CUDA_CHECK_RETURN(value)                                               \
{                                                                              \
    hipError_t _m_cudaStat = value;                                           \
    if (_m_cudaStat != hipSuccess) {                                          \
      fprintf(stderr, "Error %s at line %d in file %s\n",                      \
              hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);            \
      exit(1);                                                                 \
    }                                                                          \
}

/**
 * Função device que calcula o valor da função a ser integrada
 * @param x Coordenada x do ponto
 * @param y Coordenada y do ponto
 * @return Valor de sin(x² + y²) no ponto (x,y)
 */
__device__ double f(double x, double y) {
    return sin(x*x + y*y);
}

/**
 * Kernel CUDA que implementa a integração pelo método do trapézio
 * @param result Ponteiro para o resultado da integração (em memória de device)
 * @param nx Número de intervalos na direção x
 * @param ny Número de intervalos na direção y
 * 
 * O kernel divide o domínio de integração entre as threads disponíveis,
 * onde cada thread calcula sua contribuição local para a integral.
 * Os pontos nas bordas são tratados com peso 0.5 conforme o método do trapézio.
 */
__global__ void trapezoidalIntegration(double *result, int nx, int ny) {
    // Define os limites do domínio de integração
    const double x_start = 0.0;
    const double x_end = 1.5;
    const double y_start = 0.0;
    const double y_end = 1.5;
    
    // Calcula o tamanho dos intervalos em cada direção
    const double dx = (x_end - x_start) / nx;
    const double dy = (y_end - y_start) / ny;
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;
    double local_sum = 0.0;

    // Calcula o número total de pontos e distribui entre as threads
    long long total_points = (long long)(nx + 1) * (ny + 1);
    long long points_per_thread = (total_points + total_threads - 1) / total_threads;
    long long start_point = tid * points_per_thread;
    long long end_point = min(start_point + points_per_thread, total_points);
    
    // Itera sobre os pontos atribuídos a esta thread
    for(long long global_idx = start_point; global_idx < end_point; global_idx++) {
        // Converte índice para coordenadas 2D
        int i = global_idx / (ny + 1);
        int j = global_idx % (ny + 1);
        
        if(i <= nx && j <= ny) {
            double x = x_start + i * dx;
            double y = y_start + j * dy;
            
            // Aplica os pesos do método do trapézio para pontos nas bordas
            if(i == 0 || j == 0 || i == nx - 1 || j == ny - 1)
                local_sum += f(x, y) * 0.5f;
            else
                local_sum += f(x, y); 
        }
    }
    // Multiplica pela área do elemento de integração
    local_sum *= dx * dy;
    // Soma atomicamente ao resultado global
    atomicAdd(result, local_sum);
}

int main() {
    const int block_sizes[] = {10, 100, 1000};
    const int interval_sizes[] = {1000, 10000, 100000};

    // Itera sobre todas as combinações de parâmetros
    for(int b = 0; b < 3; b++) {
        for(int ix = 0; ix < 3; ix++) {
            for(int iy = 0; iy < 3; iy++) {
                float total_time = 0.0;
                double total_int = 0.0;
                int blocks = block_sizes[b];
                int nx = interval_sizes[ix];
                int ny = interval_sizes[iy];
                // Executa 10 vezes para média
                for(int exec = 0; exec < NUM_RUNS; exec++) {
                    // Aloca e inicializa memória na GPU
                    double *d_result;
                    CUDA_CHECK_RETURN(hipMalloc(&d_result, sizeof(double)));
                    CUDA_CHECK_RETURN(hipMemset(d_result, 0, sizeof(double)));

                    // Cria eventos para medição de tempo
                    hipEvent_t start, stop;
                    CUDA_CHECK_RETURN(hipEventCreate(&start));
                    CUDA_CHECK_RETURN(hipEventCreate(&stop));
                    
                    // Executa e mede o tempo do kernel
                    CUDA_CHECK_RETURN(hipEventRecord(start));
                    trapezoidalIntegration<<<blocks, THREADS_PER_BLOCK>>>(d_result, nx, ny);
                    CUDA_CHECK_RETURN(hipEventRecord(stop));
                    
                    // Copia o resultado de volta para CPU
                    double result;
                    CUDA_CHECK_RETURN(hipMemcpy(&result, d_result, sizeof(double), hipMemcpyDeviceToHost));
                    total_int += result;

                    // Calcula o tempo de execução
                    float milliseconds = 0;
                    CUDA_CHECK_RETURN(hipEventElapsedTime(&milliseconds, start, stop));
                    total_time += milliseconds;

                    CUDA_CHECK_RETURN(hipFree(d_result));
                    CUDA_CHECK_RETURN(hipEventDestroy(start));
                    CUDA_CHECK_RETURN(hipEventDestroy(stop));
                }

                // Calcula e imprime médias                
                float avg_time = total_time / NUM_RUNS;

                printf("%d, %d, %d, %15.15f, %15.15lf\n", 
                       blocks, nx, ny, 
                       avg_time, 
                       total_int / NUM_RUNS);
            }
        }
    }
    
    return 0;
}